#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <fstream>
#include <string>
#include <sstream>
#include <iomanip>
#include <iostream>
#include <glm/glm.hpp>

#include "common.h"
#include "cpu.h"
#include "svd3.h"

namespace ScanMatching {
	float* x_corr;
	float* R;
	float* translation;

	void findCorrespondence(float* arr1, long numArr1, float* arr2, long numArr2, float* arr1_correspondence) {
		for (int i = 0; i < numArr1 / 3; i++) {
			glm::vec3 point(arr1[i * 3 + 0], arr1[i * 3 + 1], arr1[i * 3 + 2]);
			float min_dist = LONG_MAX;
			glm::vec3 closest_point;
			for (int j = 0; j < numArr2 / 3; j++) {
				glm::vec3 other_point(arr2[j * 3 + 0], arr2[j * 3 + 1], arr2[j * 3 + 2]);
				float dist = glm::distance(point, other_point);
				if (dist < min_dist) {
					closest_point = other_point;
					min_dist = dist;
				}
			}
			arr1_correspondence[i * 3 + 0] = closest_point.x;
			arr1_correspondence[i * 3 + 1] = closest_point.y;
			arr1_correspondence[i * 3 + 2] = closest_point.z;
		}
	}

	glm::vec3 meanCenter(float* arr, float* centered, int num) {
		float meanX = 0.0f;
		float meanY = 0.0f;
		float meanZ = 0.0f;

		for (int i = 0; i < num / 3; i++) {
			meanX += arr[i * 3 + 0];
			meanY += arr[i * 3 + 1];
			meanZ += arr[i * 3 + 2];
		}

		meanX = 3.0f * meanX / num;
		meanY = 3.0f * meanY / num;
		meanZ = 3.0f * meanZ / num;

		for (int i = 0; i < num / 3; i++) {
			centered[i * 3 + 0] = arr[i * 3 + 0] - meanX;
			centered[i * 3 + 1] = arr[i * 3 + 1] - meanY;
			centered[i * 3 + 2] = arr[i * 3 + 2] - meanZ;
		}
		return glm::vec3(meanX, meanY, meanZ);
	}

	// Takes m x n matrix and returns n x m
	void transpose(float* arr, float* arrTrans, int m, int n) {
		for (int i = 0; i < m; i++) {
			for (int j = 0; j < n; j++) {
				
				arrTrans[m*j + i] = arr[n*i + j];
			}
		}
	}

	// A - m x n || B - n x p and returns C - m x p
	void multiplyMatrix(float* A, float* B, float* C, int m, int n, int p) {
		for (int i = 0; i < m; i++) {
			for (int j = 0; j < p; j++) {
				C[i*p + j] = 0;
				for (int k = 0; k < n; k++) {
					C[i*p + j] += (A[i*n + k] * B[k*p + j]);
				}
			}
		}
	}

	void subtractMatrices(float* A, float* B, float* C, int m, int n) {
		for (int i = 0; i < m; i++) {
			for (int j = 0; j < n; j++) {
				C[i*m + j] = A[i*m + j] - B[i*m + j];
			}
		}
	}

	//In-Place addition
	void addTranslation(float* A, float* trans, int num) {
		for (int i = 0; i < num; i++) {
			A[i * 3 + 0] += trans[0];
			A[i * 3 + 1] += trans[1];
			A[i * 3 + 2] += trans[2];
		}
	}

	void printMatrix(float* A, int m, int n) {
		for (int i = 0; i < m; i++) {
			for (int j = 0; j < n; j++) {
				std::cout << A[i*n + j] << " ";
			}
			std::cout << std::endl;
		}
	}

	void initScan(int numX) {
		x_corr = (float*)malloc(numX * sizeof(float));
		R = (float*)malloc(3 * 3 * sizeof(float));
		translation = (float*)malloc(3 * 1 * sizeof(float));

	}

	void match(float* x, float* y, int numX, int numY) {
		
		//Find correspondence
		

		findCorrespondence(x, numX, y, numY, x_corr);
		//std::cout << "x[0] = " << x_corr[3] << " x[1] = " << x_corr[4] << " x[2] = " << x_corr[5] << std::endl;
		

		float* x_mean_centered = (float*)malloc(numX * sizeof(float));
		float* x_corr_mean_centered = (float*)malloc(numX * sizeof(float));

		
		//Mean center x and x_corr
		glm::vec3 x_mean = meanCenter(x, x_mean_centered, numX);
	
		glm::vec3 x_corr_mean = meanCenter(x_corr, x_corr_mean_centered, numX);
		

		
		//Transpose X_corr
		float* x_corr_tr = (float*)malloc(numX * sizeof(float));
		transpose(x_corr_mean_centered, x_corr_tr, numX / 3, 3);

		

		
		//Compute C_corr_tr x X
		float* to_svd = (float*)malloc(3 * 3 * sizeof(float));
		multiplyMatrix(x_corr_tr, x_mean_centered, to_svd, 3, numX / 3, 3);

		
		
		float* svd_u = (float*)malloc(3 * 3 * sizeof(float));
		memset(svd_u, 0.0f, 3 * 3 * sizeof(float));
		float* svd_v = (float*)malloc(3 * 3 * sizeof(float));
		memset(svd_v, 0.0f, 3 * 3 * sizeof(float));
		float* svd_s = (float*)malloc(3 * 3 * sizeof(float));
		memset(svd_s, 0.0f, 3 * 3 * sizeof(float));

		svd(to_svd[0], to_svd[1], to_svd[2], to_svd[3], to_svd[4], to_svd[5], to_svd[6], to_svd[7], to_svd[8],
			svd_u[0], svd_u[1], svd_u[2], svd_u[3], svd_u[4], svd_u[5], svd_u[6], svd_u[7], svd_u[8],
			svd_s[0], svd_s[1], svd_s[2], svd_s[3], svd_s[4], svd_s[5], svd_s[6], svd_s[7], svd_s[8],
			svd_v[0], svd_v[1], svd_v[2], svd_v[3], svd_v[4], svd_v[5], svd_v[6], svd_v[7], svd_v[8]);

		
		
		//Compute U x V_tr to get rotation matrix
		float* v_tr = (float*)malloc(3 * 3 * sizeof(float));
		transpose(svd_v, v_tr, 3, 3);
		

		multiplyMatrix(svd_u, v_tr, R, 3, 3, 3);
		

		float* R_tr = (float*)malloc(9 * sizeof(float));
		transpose(R, R_tr, 3, 3);

		//Compute R x X_mean
		float* inter = (float*)malloc(3 * 1 * sizeof(float));
		float x_mean_arr[] = { x_mean.x, x_mean.y, x_mean.z };

		

		multiplyMatrix(R, x_mean_arr, inter, 3, 3, 1);

		
		//Compute Translation = x_corr_mean - R * x_mean
		float y_mean_arr[] = { x_corr_mean.x, x_corr_mean.y, x_corr_mean.z };

		
		printMatrix(y_mean_arr, 1, 3);
		std::cout << std::endl;

		

		subtractMatrices(y_mean_arr, inter, translation, 1, 3);
		

		

		//Apply the rotation matrix to current X
		float* newX = (float*)malloc(numX * sizeof(float));
		multiplyMatrix(x, R_tr, newX, numX/3 , 3, 3);

		//Add translation to every vertes
		addTranslation(newX, translation, numX / 3);
		

		//Copy updated X back
		memcpy(x, newX, numX * sizeof(float));
	}
	
}
