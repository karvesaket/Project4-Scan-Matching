#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include "common.h"
#include "naivegpu.h"
#include ""
#include <fstream>
#include <glm/glm.hpp>
#include <hipblas.h>
#include <thrust\host_vector.h>
#include <thrust\device_vector.h>
#include <thrust\reduce.h>
#include "svd3.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

#define blockSize 128


__global__ void findCorrespondence(float* arr1, long numArr1, float* arr2, long numArr2, float* arr1_correspondence) {
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= (numArr1/3)) {
		return;
	}
	glm::vec3 point(arr1[index * 3 + 0], arr1[index * 3 + 1], arr1[index * 3 + 2]);
	float min_dist = LONG_MAX;
	glm::vec3 closest_point;
	for (int j = 0; j < numArr2 / 3; j++) {
		glm::vec3 other_point(arr2[j * 3 + 0], arr2[j * 3 + 1], arr2[j * 3 + 2]);
		float dist = glm::distance(point, other_point);
		if (dist < min_dist) {
			closest_point = other_point;
			min_dist = dist;
		}
	}
	arr1_correspondence[index * 3 + 0] = closest_point.x;
	arr1_correspondence[index * 3 + 1] = closest_point.y;
	arr1_correspondence[index * 3 + 2] = closest_point.z;
}

__global__ void transpose(float* arr, float* arrTrans, int m, int n) {
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= m*n) {
		return;
	}
	int i = index / n;
	int j = index % n;

	arrTrans[m*j + i] = arr[n*i + j];
}


// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(hipblasHandle_t &handle, const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda = m, ldb = k, ldc = m;
	 float alf = 1;
	 
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

__global__ void matrix_subtraction(float* A, float* B, float* C, int m, int n) {
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= m*n) {
		return;
	}
	C[index] = A[index] - B[index];
}

__global__ void kernMatrixMultiply(float *dev_A, float *dev_B, float *dev_C, int m, int n, int k) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	double sum = 0;
	if (col < k && row < m)
	{
		for (int i = 0; i < n; i++)
			sum += dev_A[row * n + i] * dev_B[i * k + col];
		dev_C[row * k + col] = sum;
	}
}

// A - m x n || B - n x p and returns C - m x p
void multiplyMatrixCPU(float* A, float* B, float* C, int m, int n, int p) {
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < p; j++) {
			C[i*p + j] = 0;
			for (int k = 0; k < n; k++) {
				C[i*p + j] += (A[i*n + k] * B[k*p + j]);
			}
		}
	}
}


__global__ void addTranslation(float* A, float* trans, int num) {
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= num) {
		return;
	}
	A[index * 3 + 0] += trans[0];
	A[index * 3 + 1] += trans[1];
	A[index * 3 + 2] += trans[2];
}

__global__ void upSweepOptimized(int n, int d, float* A) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);


	int other_index = 1 << d;
	int stride = other_index * 2;

	int new_index = stride * index;
	if (new_index >= n) {
		return;
	}
	A[new_index + stride - 1] += A[new_index + other_index - 1];
}

__global__ void meanCenter(float* arr, float* centered, int num, float mx, float my, float mz) {
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= num) {
		return;
	}
	centered[index * 3 + 0] = arr[index * 3 + 0] - mx;
	centered[index * 3 + 1] = arr[index * 3 + 1] - my;
	centered[index * 3 + 2] = arr[index * 3 + 2] - mz;
}

__global__ void setValueOnDevice(float* device_var, float val) {
	*device_var = val;
}

//__global__ void get_svd(float* input, float* u, float* s, float* v) {
//	svd(input[0], input[1], input[2], input[3], input[4], input[5], input[6], input[7], input[8],
//		u[0], u[1], u[2], u[3], u[4], u[5], u[6], u[7], u[8],
//		s[0], s[1], s[2], s[3], s[4], s[5], s[6], s[7], s[8],
//		v[0], v[1], v[2], v[3], v[4], v[5], v[6], v[7], v[8]);
//}

__global__ void divide_sum_to_mean(float* sum, int num) {
	(*sum) = (*sum) / num;
}

void getArraySum(int n, float* input, float* sum) {
	float* padded_idata;
	int padded_size = 1 << (ilog2ceil(n));

	hipMalloc((void**)&padded_idata, padded_size * sizeof(float));
	checkCUDAErrorWithLine("hipMalloc padded_idata failed!");

	hipMemset(padded_idata, 0, padded_size * sizeof(float));
	hipMemcpy(padded_idata, input, sizeof(float) * n, hipMemcpyDeviceToDevice);

	int iterations = ilog2(padded_size);

	int number_of_threads = padded_size;
	for (int d = 0; d < iterations; d++) {
		number_of_threads /= 2;
		dim3 fullBlocksPerGridUpSweep((number_of_threads + blockSize - 1) / blockSize);
		upSweepOptimized << <fullBlocksPerGridUpSweep, blockSize >> >(padded_size, d, padded_idata);
	}

	hipMemcpy(sum, padded_idata + padded_size - 1, sizeof(float), hipMemcpyDeviceToDevice);

	hipFree(padded_idata);
}

void printMatrix(float* A, int m, int n) {
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			std::cout << A[i*n + j] << " ";
		}
		std::cout << std::endl;
	}
}

namespace NaiveGPU {
	float* dev_x;
	float* dev_y;

	float* dev_x_corr;
	float* dev_R;
	float* dev_translation;


	void initScan(int numX) {
		hipMalloc((void**)&dev_x_corr, numX * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_x_corr failed!");

		hipMalloc((void**)&dev_R, 3 * 3 * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_R failed!");

		hipMalloc((void**)&dev_translation, 3 * 1 * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_translation failed!");
	}

	void match(float* x, float* y, int numX, int numY) {

		int eachX = numX / 3;
		int eachY = numY / 3;

		dim3 numBlocks((eachX + blockSize - 1) / blockSize);
		dim3 numBlocks1((numX+blockSize - 1) / blockSize);
		dim3 numBlocks2((3 * 3 + blockSize - 1) / blockSize);
		dim3 numBlocks3((3 * 1 + blockSize - 1) / blockSize);

		dim3 dimBlock(16, 16);
		dim3 dimGrid;

		//Copy data to GPU
		hipMalloc((void**)&dev_x, numX * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_x failed!");
		hipMemcpy(dev_x, x, sizeof(float) * numX, hipMemcpyHostToDevice);

		hipMalloc((void**)&dev_y, numY * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_y failed!");
		hipMemcpy(dev_y, y, sizeof(float) * numY, hipMemcpyHostToDevice);

		
		//Find correspondence
		
		//Find Correspondence
		findCorrespondence << <numBlocks, blockSize >> >(dev_x, numX, dev_y, numY, dev_x_corr);

		//Transpose x_corr and x
		float* dev_x_tr;
		hipMalloc((void**)&dev_x_tr, numX * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_x failed!");
		transpose << <numBlocks1, blockSize >> >(dev_x, dev_x_tr, eachX, 3);

		float* dev_x_corr_tr;
		hipMalloc((void**)&dev_x_corr_tr, numX * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_x failed!");
		transpose << <numBlocks1, blockSize >> >(dev_x_corr, dev_x_corr_tr, eachX, 3);

		
		float meanX;
		meanX = thrust::reduce(thrust::device, dev_x_tr, dev_x_tr + eachX, 0.0f);
		meanX /= eachX;
		
		float meanY;
		meanY = thrust::reduce(thrust::device, dev_x_tr + eachX, dev_x_tr + (2 * eachX), 0.0f);
		meanY /= eachX;
		
		float meanZ;
		meanZ = thrust::reduce(thrust::device, dev_x_tr + (2 * eachX), dev_x_tr + numX, 0.0f);
		meanZ /= eachX;
		

		
		
		hipFree(dev_x_tr);

		
		//Mean-center x_corr
		float meanXC;
		meanXC = thrust::reduce(thrust::device, dev_x_corr_tr, dev_x_corr_tr + eachX, 0.0f);
		meanXC /= eachX;
		
		float meanYC;
		meanYC = thrust::reduce(thrust::device, dev_x_corr_tr + eachX, dev_x_corr_tr + (2 * eachX), 0.0f);
		meanYC /= eachX;
		
		float meanZC;
		meanZC = thrust::reduce(thrust::device, dev_x_corr_tr + (2 * eachX), dev_x_corr_tr + numX, 0.0f);
		meanZC /= eachX;
		

		

		hipFree(dev_x_corr_tr);

		float* dev_x_mean_center;
		hipMalloc((void**)&dev_x_mean_center, numX * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_x failed!");
		

		float* dev_x_corr_mean_center;
		hipMalloc((void**)&dev_x_corr_mean_center, numX * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_y failed!");
		

		
		meanCenter <<<numBlocks, blockSize >>>(dev_x, dev_x_mean_center, eachX, meanX, meanY, meanZ);
		
		meanCenter <<<numBlocks, blockSize >>>(dev_x_corr, dev_x_corr_mean_center, eachX, meanXC, meanYC, meanZC);

		
		//Multiply x_corr_tr and x to get input to SVD
		hipMalloc((void**)&dev_x_corr_tr, numX * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_x failed!");
		transpose << <numBlocks1, blockSize >> > (dev_x_corr_mean_center, dev_x_corr_tr, eachX, 3);

		
		

		float* dev_to_svd;
		hipMalloc((void**)&dev_to_svd, 3 * 3 * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_to_svd failed!");

		float* to_svd = (float*)malloc(3 * 3 * sizeof(float));

		//multiplyMatrixCPU(y_tr_cpu, x_cpu, to_svd, 3, eachX, 3);

		//hipMemcpy(dev_to_svd, to_svd, sizeof(float) * 9, hipMemcpyHostToDevice);

		dimGrid.x = (3 + dimBlock.x - 1) / dimBlock.x;
		dimGrid.y = (3 + dimBlock.y - 1) / dimBlock.y;
		kernMatrixMultiply << <dimGrid, dimBlock >> > (dev_x_corr_tr, dev_x_mean_center, dev_to_svd, 3, eachX, 3);

		// Create a handle for CUBLAS
		hipblasHandle_t handle;
		hipblasCreate(&handle);
		//gpu_blas_mmul(handle, dev_x_corr_tr, dev_x, dev_to_svd, 3, eachX, 3);
		//multiplyMatrix<<<numBlocks2, blockSize>> > (dev_x_corr_tr, dev_x, dev_to_svd, 3, eachX, 3);

		
		hipMemcpy(to_svd, dev_to_svd, sizeof(float) * 9, hipMemcpyDeviceToHost);
		

		float* svd_u = (float*)malloc(3 * 3 * sizeof(float));
		memset(svd_u, 0.0f, 3 * 3 * sizeof(float));
		float* svd_v = (float*)malloc(3 * 3 * sizeof(float));
		memset(svd_v, 0.0f, 3 * 3 * sizeof(float));
		float* svd_s = (float*)malloc(3 * 3 * sizeof(float));
		memset(svd_s, 0.0f, 3 * 3 * sizeof(float));

		svd(to_svd[0], to_svd[1], to_svd[2], to_svd[3], to_svd[4], to_svd[5], to_svd[6], to_svd[7], to_svd[8],
			svd_u[0], svd_u[1], svd_u[2], svd_u[3], svd_u[4], svd_u[5], svd_u[6], svd_u[7], svd_u[8],
			svd_s[0], svd_s[1], svd_s[2], svd_s[3], svd_s[4], svd_s[5], svd_s[6], svd_s[7], svd_s[8],
			svd_v[0], svd_v[1], svd_v[2], svd_v[3], svd_v[4], svd_v[5], svd_v[6], svd_v[7], svd_v[8]);

		
		//Find SVD - U, V, S
		float* dev_svd_u;
		hipMalloc((void**)&dev_svd_u, 3 * 3 * sizeof(float));
		hipMemset(dev_svd_u, 0.0f, 3 * 3 * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_to_svd failed!");

		float* dev_svd_s;
		hipMalloc((void**)&dev_svd_s, 3 * 3 * sizeof(float));
		hipMemset(dev_svd_s, 0.0f, 3 * 3 * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_to_svd failed!");

		float* dev_svd_v;
		hipMalloc((void**)&dev_svd_v, 3 * 3 * sizeof(float));
		hipMemset(dev_svd_v, 0.0f, 3 * 3 * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_to_svd failed!");

		//get_svd << <1, 1 >> > (dev_to_svd, dev_svd_u, dev_svd_s, dev_svd_v);

		//float* u = (float*)malloc(3 * 3 * sizeof(float));
		hipMemcpy(dev_svd_u, svd_u, sizeof(float) * 9, hipMemcpyHostToDevice);
		

		//float* v = (float*)malloc(3 * 3 * sizeof(float));
		hipMemcpy(dev_svd_v, svd_v, sizeof(float) * 9, hipMemcpyHostToDevice);
		

		
		hipFree(dev_svd_s);
		//Compute U x V_tr to get R
		float* dev_svd_v_tr;
		hipMalloc((void**)&dev_svd_v_tr, 3 * 3 * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_to_svd failed!");
		transpose << <numBlocks2, blockSize >> > (dev_svd_v, dev_svd_v_tr, 3, 3);

		float* v_tr = (float*)malloc(3 * 3 * sizeof(float));
		hipMemcpy(v_tr, dev_svd_v_tr, sizeof(float) * 9, hipMemcpyDeviceToHost);
		

		hipFree(dev_svd_v);
		

		
		dimGrid.x = (3 + dimBlock.x - 1) / dimBlock.x;
		dimGrid.y = (3 + dimBlock.y - 1) / dimBlock.y;
		kernMatrixMultiply << <dimGrid, dimBlock >> > (dev_svd_u, dev_svd_v_tr, dev_R, 3, 3, 3);

		//gpu_blas_mmul(handle, dev_svd_u, dev_svd_v_tr, dev_R, 3, 3, 3);

		float* R = (float*)malloc(3 * 3 * sizeof(float));
		hipMemcpy(R, dev_R, sizeof(float) * 9, hipMemcpyDeviceToHost);
		

		
		//Compute translation = x_corr_mean - R.x_mean
		float* dev_x_mean;
		hipMalloc((void**)&dev_x_mean, 3 * sizeof(float));
		setValueOnDevice << <1, 1 >> > (&dev_x_mean[0], meanX);
		setValueOnDevice << <1, 1 >> > (&dev_x_mean[1], meanY);
		setValueOnDevice << <1, 1 >> > (&dev_x_mean[2], meanZ);

		float* dev_y_mean;
		hipMalloc((void**)&dev_y_mean, 3 * sizeof(float));
		setValueOnDevice << <1, 1 >> > (&dev_y_mean[0], meanXC);
		setValueOnDevice << <1, 1 >> > (&dev_y_mean[1], meanYC);
		setValueOnDevice << <1, 1 >> > (&dev_y_mean[2], meanZC);

		

		float* dev_R_tr;
		hipMalloc((void**)&dev_R_tr, 9 * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_x failed!");
		transpose << <numBlocks2, blockSize >> > (dev_R, dev_R_tr, 3, 3);

		float* inter;
		hipMalloc((void**)&inter, 3 * 1 * sizeof(float));
		dimGrid.x = (3 + dimBlock.x - 1) / dimBlock.x;
		dimGrid.y = (1 + dimBlock.y - 1) / dimBlock.y;
		kernMatrixMultiply << <dimGrid, dimBlock >> > (dev_R, dev_x_mean, inter, 3, 3, 1);
		//gpu_blas_mmul(handle, dev_R, dev_x_mean, inter, 3, 3, 1);

		matrix_subtraction << <numBlocks3, blockSize >> > (dev_y_mean, inter, dev_translation, 1, 3);

		float* trans = (float*)malloc(3 * 1 * sizeof(float));
		hipMemcpy(trans, dev_translation, sizeof(float) * 3, hipMemcpyDeviceToHost);
		

		
		
		//Apply rotation on x
		float* dev_newX;
		hipMalloc((void**)&dev_newX, numX * sizeof(float));
		dimGrid.x = (3 + dimBlock.x - 1) / dimBlock.x;
		dimGrid.y = (eachX + dimBlock.y - 1) / dimBlock.y;
		kernMatrixMultiply << <dimGrid, dimBlock >> > (dev_x, dev_R_tr, dev_newX, eachX, 3, 3);
		//gpu_blas_mmul(handle, dev_x, dev_R, dev_newX, eachX, 3, 3);
		hipDeviceSynchronize();

		//Apply translation on x
		addTranslation << <numBlocks, blockSize >> > (dev_newX, dev_translation, eachX);

		hipMemcpy(x, dev_newX, sizeof(float) * numX, hipMemcpyDeviceToHost);
		
		hipDeviceSynchronize();
	}
}

